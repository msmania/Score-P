#include "hip/hip_runtime.h"
/*
 * This file is part of the Score-P software (http://www.score-p.org)
 *
 * Copyright (c) 2009-2013,
 * RWTH Aachen University, Germany
 *
 * Copyright (c) 2009-2013,
 * Gesellschaft fuer numerische Simulation mbH Braunschweig, Germany
 *
 * Copyright (c) 2009-2013, 2015,
 * Technische Universitaet Dresden, Germany
 *
 * Copyright (c) 2009-2013,
 * University of Oregon, Eugene, USA
 *
 * Copyright (c) 2009-2013,
 * Forschungszentrum Juelich GmbH, Germany
 *
 * Copyright (c) 2009-2013,
 * German Research School for Simulation Sciences GmbH, Juelich/Aachen, Germany
 *
 * Copyright (c) 2009-2013,
 * Technische Universitaet Muenchen, Germany
 *
 * This software may be modified and distributed under the terms of
 * a BSD-style license. See the COPYING file in the package base
 * directory for details.
 *
 */


/**
 * @file
 *
 * @brief Test program for the CUDA adapter. Several parts of this program have
 * been extracted from the NVIDIA computing samples 'simpleStreams' and
 * 'concurrentKernels'
 *
 * The basic test runs one kernel in (1+num_streams) streams.
 *
 * This advanced test runs (1+nreps*num_streams) instances of kernel 'init_array' and
 * (num_streams) instances of kernel 'clock_block'.
 */

#include <stdio.h>
#include <unistd.h>
#include <stdint.h>

#include <SCOREP_User.h>

// CUDA utilities and system includes
#include <hip/hip_runtime.h>

#define CUDART_CALL( _err, _msg ) \
    if ( hipSuccess != _err )         \
        __checkCUDACall( _err, _msg, __FILE__, __LINE__ )

static uint32_t kernel_workload = 20;

static uint64_t cpu_usleeptime = 10000;

static uint32_t num_streams = 3;

static int basic_mode = 0;

/* function declarations */
static void
__checkCUDACall( hipError_t ecode,
                 const char* msg,
                 const char* file,
                 const int   line );

static void
runBasicTest( int nstreams );

static void
runCopyComputeOverlap( int nstreams );

static void
runConcurrentKernels( int nstreams );

static void
show_help( void );

static char
getopt( char* argument );

static void setArguments( int argc, char* argv[] );


__global__ void
init_array( int* g_data, int* factor, int num_iterations )
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for ( int i = 0; i < num_iterations; i++ )
    {
        g_data[ idx ] += *factor; // non-coalesced on purpose, to burn time
    }
}

// This is a kernel that does no real work but runs at least for a specified number of clocks
__global__ void
clock_block( clock_t* d_o, clock_t clock_count )
{
    unsigned int start_clock = ( unsigned int )clock();

    clock_t clock_offset = 0;

    while ( clock_offset < clock_count )
    {
        unsigned int end_clock = ( unsigned int )clock();

        // The code below should work like
        // this (thanks to modular arithmetics):
        //
        // clock_offset = (clock_t) (end_clock > start_clock ?
        //                           end_clock - start_clock :
        //                           end_clock + (0xffffffffu - start_clock));
        //
        // Indeed, let m = 2^32 then
        // end - start = end + m - start (mod m).

        clock_offset = ( clock_t )( end_clock - start_clock );
    }

    d_o[ 0 ] = clock_offset;
}

int
main( int argc, char** argv )
{
    SCOREP_USER_FUNC_BEGIN()

    // check the compute capability of the device
    int num_devices = 0;

    setArguments( argc, argv );

    CUDART_CALL( hipGetDeviceCount( &num_devices ), "hipGetDeviceCount" );
    if ( 0 == num_devices )
    {
        printf( "your system does not have a CUDA capable device, waiving test...\n" );
        SCOREP_USER_FUNC_END()
        exit( 77 ); /* denote the test as skipped */
    }

    /* check if the command-line chosen device ID is within range, exit if not
       if( cuda_device >= num_devices ){
       printf("cuda_device=%d is invalid, must choose device ID between 0 and %d\n", cuda_device, num_devices-1);
       SCOREP_USER_FUNC_END()
       exit(-1);
       }*/

    if ( basic_mode )
    {
        runBasicTest( num_streams );
    }
    else
    {
        runCopyComputeOverlap( num_streams );

        runConcurrentKernels( num_streams );
    }

    hipDeviceReset();

    SCOREP_USER_FUNC_END()
}

static void
runBasicTest( int nstreams )
{
    SCOREP_USER_FUNC_BEGIN()

    int n = 512 * 1024;                 // number of integers in the data set
    int  nbytes = n * sizeof( int );    // number of data bytes
    dim3 threads, blocks;               // kernel launch configuration
    int  niterations = kernel_workload; // number of iterations for the loop inside the kernel_time

    // allocate host memory
    int  c   = 5;               // value to which the array will be initialized
    int* h_a = 0;               // pointer to the array data in host memory

    printf( "Starting basic test\n" );
    h_a = ( int* )malloc( nbytes );

    // allocate device memory
    int* d_a = 0, * d_c = 0;         // pointers to data and init value in the device memory
    CUDART_CALL( hipMalloc( ( void** )&d_a, nbytes ), "hipMalloc" );
    CUDART_CALL( hipMalloc( ( void** )&d_c, sizeof( int ) ), "hipMalloc" );

    CUDART_CALL( hipMemset( d_a, 0, nbytes ), "hipMemset" );
    CUDART_CALL( hipMemcpy( d_c, &c, sizeof( int ), hipMemcpyHostToDevice ), "hipMemcpy" );

    threads = dim3( 512, 1 );
    blocks  = dim3( n / threads.x, 1 );
    hipStream_t* streams = ( hipStream_t* )malloc( nstreams * sizeof( hipStream_t ) );
    for ( int i = 0; i < nstreams; i++ )
    {
        CUDART_CALL( hipStreamCreate( &( streams[ i ] ) ), "hipStreamCreate" );
    }

    SCOREP_USER_REGION_BY_NAME_BEGIN( "init_array", SCOREP_USER_REGION_TYPE_COMMON )
    init_array<<<blocks, threads>>>( d_a, d_c, niterations );
    SCOREP_USER_REGION_BY_NAME_END( "init_array" )
    CUDART_CALL( hipDeviceSynchronize(), "hipDeviceSynchronize" );

    for ( int i = 0; i < nstreams; i++ )
    {
        SCOREP_USER_REGION_BY_NAME_BEGIN( "init_array", SCOREP_USER_REGION_TYPE_COMMON )
        init_array<<<blocks, threads, 0, streams[ i ]>>>( d_a, d_c, niterations );
        SCOREP_USER_REGION_BY_NAME_END( "init_array" )
        CUDART_CALL( hipDeviceSynchronize(), "hipDeviceSynchronize" );
    }

    // cleanup
    for ( int i = 0; i < nstreams; i++ )
    {
        hipStreamDestroy( streams[ i ] );
    }
    free( streams );

    hipFree( d_a );
    hipFree( d_c );
    free( h_a );

    SCOREP_USER_FUNC_END()
}

static void
runCopyComputeOverlap( int nstreams )
{
    SCOREP_USER_FUNC_BEGIN()

    int nreps = 3;                      // number of times each experiment is repeated
    int  n      = 512 * 1024;           // number of integers in the data set
    int  nbytes = n * sizeof( int );    // number of data bytes
    dim3 threads, blocks;               // kernel launch configuration
    int  niterations = kernel_workload; // number of iterations for the loop inside the kernel_time

    // allocate host memory
    int  c          = 5;      // value to which the array will be initialized
    int* h_a        = 0;      // pointer to the array data in host memory
    int* hAligned_a = 0;      // pointer to the array data in host memory (aligned to MEMORY_ALIGNMENT)

    // allocate host memory (pinned is required for achieve asynchronicity)
    CUDART_CALL( hipHostMalloc( ( void** )&h_a, nbytes ), "hipHostMalloc" );
    hAligned_a = h_a;

    // allocate device memory
    int* d_a = 0, * d_c = 0;                // pointers to data and init value in the device memory
    CUDART_CALL( hipMalloc( ( void** )&d_a, nbytes ), "hipMalloc" );
    CUDART_CALL( hipMalloc( ( void** )&d_c, sizeof( int ) ), "hipMalloc" );
    CUDART_CALL( hipMemcpy( d_c, &c, sizeof( int ), hipMemcpyHostToDevice ), "hipMemcpy" );

    threads = dim3( 512, 1 );
    blocks  = dim3( n / threads.x, 1 );
    SCOREP_USER_REGION_BY_NAME_BEGIN( "init_array", SCOREP_USER_REGION_TYPE_COMMON )
    init_array<<<blocks, threads>>>( d_a, d_c, niterations );
    SCOREP_USER_REGION_BY_NAME_END( "init_array" )
    usleep( cpu_usleeptime );
    hipMemcpyAsync( hAligned_a, d_a, nbytes, hipMemcpyDeviceToHost );

    // allocate and initialize an array of stream handles
    hipStream_t* streams = ( hipStream_t* )malloc( nstreams * sizeof( hipStream_t ) );
    for ( int i = 0; i < nstreams; i++ )
    {
        CUDART_CALL( hipStreamCreate( &( streams[ i ] ) ), "hipStreamCreate" );
    }

    niterations = kernel_workload;

    printf( "Starting Copy/Compute overlap test\n" );
    threads = dim3( 512, 1 );
    blocks  = dim3( n / ( nstreams * threads.x ), 1 );
    memset( hAligned_a, 255, nbytes ); // set host memory bits to all 1s, for testing correctness
    hipMemset( d_a, 0, nbytes );      // set device memory to all 0s, for testing correctness
    for ( int k = 0; k < nreps; k++ )
    {
        // asynchronously launch nstreams kernels, each operating on its own portion of data
        for ( int i = 0; i < nstreams; i++ )
        {
            SCOREP_USER_REGION_BY_NAME_BEGIN( "init_array", SCOREP_USER_REGION_TYPE_COMMON )
            init_array<<<blocks, threads, 0, streams[ i ]>>>( d_a + i * n / nstreams, d_c, niterations );
            SCOREP_USER_REGION_BY_NAME_END( "init_array" )
        }
        // asynchronously launch nstreams memcopies.  Note that memcopy in stream x will only
        //   commence executing when all previous CUDA calls in stream x have completed
        for ( int i = 0; i < nstreams; i++ )
        {
            hipMemcpyAsync( hAligned_a + i * n / nstreams, d_a + i * n / nstreams, nbytes / nstreams, hipMemcpyDeviceToHost, streams[ i ] );
        }
    }
    CUDART_CALL( hipDeviceSynchronize(), "hipDeviceSynchronize" );

    // release resources
    for ( int i = 0; i < nstreams; i++ )
    {
        hipStreamDestroy( streams[ i ] );
    }
    free( streams );

    hipHostFree( h_a );
    hipFree( d_a );
    hipFree( d_c );

    SCOREP_USER_FUNC_END()
}

static void
runConcurrentKernels( int nstreams )
{
    SCOREP_USER_FUNC_BEGIN()

    float kernel_time = 10;                                    // time the kernel should run in ms
    hipDeviceProp_t deviceProp;
    clock_t*       a           = NULL;                         // pointer to the array data in host memory
    int            nbytes      = nstreams * sizeof( clock_t ); // number of data bytes
    int            cuda_device = 0;

    CUDART_CALL( hipGetDevice( &cuda_device ), "hipGetDevice" );

    CUDART_CALL( hipGetDeviceProperties( &deviceProp, cuda_device ), "hipGetDeviceProperties" );
    if ( ( deviceProp.concurrentKernels == 0 ) )
    {
        printf( "> GPU does not support concurrent kernel execution\n" );
        printf( "  CUDA kernel runs will be serialized\n" );
    }

    // allocate host memory
    CUDART_CALL( hipHostMalloc( ( void** )&a, nbytes ), "hipHostMalloc" );

    // allocate device memory
    clock_t* d_ac = 0;                 // pointers to data and init value in the device memory
    CUDART_CALL( hipMalloc( ( void** )&d_ac, nbytes ), "hipMalloc" );

    // allocate and initialize an array of stream handles
    hipStream_t* streams = ( hipStream_t* )malloc( nstreams * sizeof( hipStream_t ) );
    for ( int i = 0; i < nstreams; i++ )
    {
        CUDART_CALL( hipStreamCreate( &( streams[ i ] ) ), "hipStreamCreate" );
    }

    // time execution with nkernels streams
    clock_t total_clocks = 0;
    clock_t time_clocks  = kernel_time * deviceProp.clockRate;
    printf( "Starting concurrent kernel test\n" );

    // queue nkernels in separate streams and record when they are done
    for ( int i = 0; i < nstreams; ++i )
    {
        SCOREP_USER_REGION_BY_NAME_BEGIN( "clock_block", SCOREP_USER_REGION_TYPE_COMMON )
        clock_block<<<1, 1, 0, streams[ i ]>>>( &d_ac[ i ], time_clocks );
        SCOREP_USER_REGION_BY_NAME_END( "clock_block" )
        total_clocks += time_clocks;
    }

    CUDART_CALL( hipDeviceSynchronize(), "hipDeviceSynchronize" );

    // release resources
    for ( int i = 0; i < nstreams; i++ )
    {
        hipStreamDestroy( streams[ i ] );
    }
    free( streams );
    hipHostFree( a );
    hipFree( d_ac );

    SCOREP_USER_FUNC_END()
}

/*
 * Checks if a CUDA runtime API call returns successful and respectively prints
 * the error.
 *
 * @param ecode the CUDA error code
 * @param msg a message to get more detailed information about the error
 * @param the corresponding file
 * @param the line the error occurred
 */
static void
__checkCUDACall( hipError_t ecode, const char* msg,
                 const char* file, const int line )
{
    if ( msg != NULL )
    {
        printf( "[CUDART] %s",  msg );
    }
    printf( "[CUDA Error <%s>:%i] %s", file, line, hipGetErrorString( ecode ) );
}

static void
show_help( void )
{
    printf( "\ncuda_test [OPTION]\n"
            "\t-g  kernel workload as number of loop iterations (positive integer)\n"
            "\t-c  sleep time of host after first kernel launch in seconds (positive integer)\n"
            "\t-s  number of CUDA streams (positive integer)\n"
            "\t-b  run only basic test (one kernel)\n\n"
            );
}

static char
getopt( char* argument )
{
    if ( argument[ 0 ] == '-' )
    {
        return argument[ 1 ];
    }
    return 'f';
}

static void
setArguments( int argc, char* argv[] )
{
    int j = 1;

    while ( j < argc )
    {
        switch ( getopt( argv[ j ] ) )
        {
            case 'g': // number of loop iterations inside GPU kernel
                kernel_workload = atoi( argv[ ++j ] );
                break;

            case 'c': // seconds to sleep after launch of first kernel
                cpu_usleeptime = atoi( argv[ ++j ] ) * 1000;
                break;

            case 's': // number of CUDA streams to use
                num_streams = atoi( argv[ ++j ] );
                break;

            case 'b': // test only basic CUDA features
                basic_mode = 1;
                break;

            default:
                show_help();
                exit( 1 );
        }
        j++;
    }
}
