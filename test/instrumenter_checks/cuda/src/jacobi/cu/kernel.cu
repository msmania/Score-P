#include "hip/hip_runtime.h"
#include "definitions.cuh"

//Performs CFD calculation on global memory. This code does not use any advance optimization technique on GPU
// But still acheives many fold performance gain
__global__ void calculateCFD_V1( float* input,  float* output, unsigned int Ni, unsigned int Nj,
                                 float h)
{
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x; // Y - ID
    unsigned int j = blockDim.y * blockIdx.y + threadIdx.y; // X - ID

    unsigned int iPrev = i-1; // Previous Y element
    unsigned int iNext = i+1; // Next Y element

    unsigned int jPrev = j-1; //Previous X element
    unsigned int jNext = j+1; // Next X element


    unsigned int index = i * Nj + j;

    if( i > 0 && j > 0 && i < (Ni-1) && j <(Nj-1))
        output[index] = 0.25f * (input[iPrev * Nj + j] + input[iNext* Nj + j] + input[i * Nj+ jPrev]
            + input[i* Nj + jNext] - 4*h*h);
}

//This version of Kernel uses optimization by copying the data into shared memory and hence results in better performance
__global__ void calculateCFD_V2( float* input,  float* output, unsigned int Ni, unsigned int Nj,
                                 float h)
{

    //Current Global ID
    int i = blockDim.y * blockIdx.y + threadIdx.y; // Y - ID
    int j = blockDim.x * blockIdx.x + threadIdx.x; // X - ID

    //Current Local ID (lXX --> refers to local ID i.e. inside a block)
    int li = threadIdx.y;
    int lj = threadIdx.x;

    // e_XX --> variables refers to expanded shared memory location in order to accomodate halo elements
    //Current Local ID with radius offset.
    int e_li = li + RADIUS;
    int e_lj = lj + RADIUS;

    // Variable pointing at top and bottom neighbouring location
    int e_li_prev = e_li - 1;
    int e_li_next = e_li + 1;

    // Variable pointing at left and right neighbouring location
    int e_lj_prev = e_lj - 1;
    int e_lj_next = e_lj + 1;

    __shared__ float sData [THREADS_PER_BLOCK_Y + 2 * RADIUS][THREADS_PER_BLOCK_X + 2 * RADIUS];

    unsigned int index = (i)* Nj + (j) ;

    if( li<RADIUS ) // copy top and bottom halo
    {
        //Copy Top Halo Element
        if(blockIdx.y > 0) // Boundary check
            sData[li][e_lj] = input[index - RADIUS * Nj];

        //Copy Bottom Halo Element
        if(blockIdx.y < (gridDim.y-1)) // Boundary check
            sData[e_li+THREADS_PER_BLOCK_Y][e_lj] = input[index + THREADS_PER_BLOCK_Y * Nj];

    }

    if( lj<RADIUS ) // copy left and right halo
    {
        if( blockIdx.x > 0) // Boundary check
            sData[e_li][lj] = input[index - RADIUS];

        if(blockIdx.x < (gridDim.x-1)) // Boundary check
            sData[e_li][e_lj+THREADS_PER_BLOCK_X] = input[index + THREADS_PER_BLOCK_X];
    }

    // copy current location
    sData[e_li][e_lj] = input[index];

    __syncthreads( );

    if( i > 0 && j > 0 && i < (Ni-1) && j <(Nj-1))
        output[index] = 0.25f * (sData[e_li_prev][e_lj] + sData[e_li_next][e_lj] + sData[e_li][e_lj_prev]
            + sData[e_li][e_lj_next] - 4*h*h);

}
